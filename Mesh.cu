#include "hip/hip_runtime.h"
#include "Mesh.cuh"
#include "book.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <unordered_map>
#include <map>

using namespace std;

Mesh::Mesh()
{
	edges = NULL; angles = NULL;
	edgeNum = 0; vertNum = 0;
}

Mesh::~Mesh()
{

}

bool Mesh::LoadFromFile(const char *fileName)
{
	ifstream input(fileName);
	if (!input)
	{
		cout << "Cannot open file " << fileName << endl;
		return false;
	}

	string curLine;
	char t;
	Vertex point;
	unsigned v0, v1, v2;
	stringstream sin;
	vector<Vertex> vertsVec;
	vector<int> facesVec;
	vector<Edge> edgesVec;

	while (getline(input, curLine))
	{
		sin.clear();
		if (curLine[0] == 'v')
		{
			if (curLine[1] == ' ' || curLine[1] == '\t')
			{
				sin << curLine;
				sin >> t >> point.pos.x >> point.pos.y >> point.pos.z;
				vertsVec.push_back(point);
			}
		}
		else if (curLine[0] == 'f')
		{
			sin << curLine;
			sin >> t >> v0 >> v1 >> v2;
			facesVec.push_back(v0 - 1); facesVec.push_back(v1 - 1); facesVec.push_back(v2 - 1);
		}
	}
	
	edgesVec.resize(facesVec.size());

	int edgeIdx = 0;
	map< pair<int, int>, int > verts2edge;

	for (int i = 0; i < facesVec.size(); i+=3)
	{
		for (int j = 0; j < 3; ++j)
		{
			int v0 = facesVec[i + j], v1 = facesVec[i + (j + 1) % 3];
			edgesVec[edgeIdx].verts[0] = v0; edgesVec[edgeIdx].verts[1] = v1;
			edgesVec[edgeIdx].edgeLen = (vertsVec[v0].pos - vertsVec[v1].pos).length();
			edgesVec[edgeIdx].nextEdge = i + (j + 1) % 3;
			edgesVec[edgeIdx].prevEdge = i + (j + 2) % 3;
			edgesVec[edgeIdx].faceId = i / 3;

			if (v0 > v1) std::swap(v0, v1);
			auto iter = verts2edge.find(make_pair(v0, v1));
			if (iter != verts2edge.end())
			{
				edgesVec[edgeIdx].twinEdge = iter->second;
				edgesVec[iter->second].twinEdge = edgeIdx;
			}
			else
			{
				verts2edge[make_pair(v0, v1)] = edgeIdx;
			}
			++edgeIdx;
		}
	}

	edgeNum = edgesVec.size();
	edges = new Edge[edgeNum];
	copy(edgesVec.begin(), edgesVec.end(), edges);

	vertNum = vertsVec.size();
	verts = new Vertex[vertNum];
	copy(vertsVec.begin(), vertsVec.end(), verts);

	angles = new double[vertNum];
	edgeAdjToVert = new int[vertNum];

	memset(angles, 0, vertNum * sizeof(double));

	faceNum = facesVec.size() / 3;
	faces = new Face[faceNum];

	for (int i = 0; i < edgeNum; ++i)
	{
		double l0 = edges[i].edgeLen;
		double l1 = edges[edges[i].nextEdge].edgeLen;
		double l2 = edges[edges[i].prevEdge].edgeLen;

		double curAngle = (l0*l0 + l2*l2 - l1*l1) / (2.0 * l0*l2);
		if (curAngle > 1.0) curAngle = 1.0; else if (curAngle < -1.0) curAngle = -1.0;
		curAngle = acos(curAngle);
		angles[edges[i].verts[0]] += curAngle;
		/*
		curAngle = (l0*l0 + l1*l1 - l2*l2) / (2.0 * l0*l1);
		if (curAngle > 1.0) curAngle = 1.0; else if (curAngle < -1.0) curAngle = -1.0;
		curAngle = acos(curAngle);
		angles[edges[i].verts[1]] += curAngle;
		*/
		edgeAdjToVert[edges[i].verts[0]] = i;
	}

	for (int i = 0; i < faceNum; ++i)
	{
		faces[i].edges[0] = i * 3 + 0;
		faces[i].edges[1] = i * 3 + 1;
		faces[i].edges[2] = i * 3 + 2;
	}

	return true;
}

bool Mesh::copyToGPU(Mesh *d_mesh)
{
	hipError_t cudaStatus;
	d_mesh->edgeNum = edgeNum;
	d_mesh->vertNum = vertNum;
	d_mesh->faceNum = faceNum;
	HANDLE_ERROR(hipMalloc((void**)&(d_mesh->edges), edgeNum * sizeof(Edge)));
	HANDLE_ERROR(hipMalloc((void**)&(d_mesh->angles), vertNum * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&(d_mesh->edgeAdjToVert), vertNum * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&(d_mesh->verts), vertNum * sizeof(Vertex)));
	HANDLE_ERROR(hipMalloc((void**)&(d_mesh->faces), faceNum * sizeof(Face)));

	HANDLE_ERROR(hipMemcpy(d_mesh->edges, edges, edgeNum * sizeof(Edge), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_mesh->angles, angles, vertNum * sizeof(double), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_mesh->edgeAdjToVert, edgeAdjToVert, vertNum * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_mesh->verts, verts, vertNum * sizeof(Vertex), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_mesh->faces, faces, faceNum * sizeof(Face), hipMemcpyHostToDevice));
	return true;
}

void Mesh::clear()
{
	if (edges) delete[] edges;
	if (angles) delete[] angles;
	if (edgeAdjToVert) delete[] edgeAdjToVert;
	if (verts) delete[] verts;
	if (faces) delete[] faces;

	edgeNum = 0; vertNum = 0; faceNum = 0;
}

void Mesh::clearGPU()
{
	HANDLE_ERROR(hipFree(edges));
	HANDLE_ERROR(hipFree(angles));
	HANDLE_ERROR(hipFree(edgeAdjToVert));
	HANDLE_ERROR(hipFree(verts));
	HANDLE_ERROR(hipFree(faces));
	edgeNum = 0; vertNum = 0; faceNum = 0;
}