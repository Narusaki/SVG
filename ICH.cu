#include "hip/hip_runtime.h"
// ICH.cpp : Defines the exported functions for the DLL application.
//

#include "ICH.cuh"
#include <iostream>
#include <list>

using namespace std;

// This is the constructor of a class that has been exported.
// see ICH.h for the class definition
__host__ __device__ ICH::ICH()
{
	sourceVert = -1; sourcePointFace = -1;

	mesh = NULL;
	storedWindows = NULL; keptFaces = NULL;
	storedWindowsIdx = 0; keptFacesIdx = 0;

	numOfWinGen = 0;
	maxWinQSize = 0;
	maxPseudoQSize = 0;
	totalCalcVertNum = 0;
	return;
}

__host__ __device__ ICH::~ICH()
{
	return;
}

__host__ __device__ void ICH::AssignMesh(Mesh *mesh_)
{
	mesh = mesh_;
}

__host__ __device__ void ICH::AssignBuffers(SplitItem *splitInfos_, unsigned splitInfoSize,
	VertItem *vertInfos_, unsigned vertInfoSize,
	PriorityQueues< Window > winQ_, PriorityQueues< PseudoWindow > pseudoSrcQ_,
	Window* storedWindows_, unsigned *keptFaces_)
{
	splitInfos = MyHashTable<SplitInfo>(splitInfoSize, splitInfos_);
	vertInfos = MyHashTable<VertInfo>(vertInfoSize, vertInfos_);
	winQ = winQ_;
	pseudoSrcQ = pseudoSrcQ_;
	storedWindows = storedWindows_;
	keptFaces = keptFaces_;
}

__host__ __device__ void ICH::AddSource(unsigned vertId)
{
	sourceVert = vertId;
}

__host__ __device__ void ICH::AddSource(unsigned faceId, Vector3D pos)
{
	sourcePointFace = faceId;
	sourcePointPos = pos;
}

__host__ __device__ void ICH::AddFacesKeptWindow(unsigned faceId)
{
	if (keptFacesIdx == keptFacesSize) return;
	keptFaces[keptFacesIdx++] = faceId;
}

__host__ __device__ void ICH::Execute(int totalCalcVertNum_)
{
	// Initialize
	Initialize();

	while (!winQ.empty() || !pseudoSrcQ.empty())
	{
		// Get valid window (for window whose pseudoSrcBirthTime is not equal (which means smaller/older) than
		// the current one, it must be an old window, which can be safely skipped)
		/*cout << "\r" << winQ.size() << " " << pseudoSrcQ.size();*/
		maxWinQSize = max(maxWinQSize, winQ.size());
		maxPseudoQSize = max(maxPseudoQSize, pseudoSrcQ.size());

		while (!winQ.empty() && winQ.top().pseudoSrcId < mesh->vertNum &&
			winQ.top().pseudoSrcBirthTime != vertInfos.get(winQ.top().pseudoSrcId).birthTime)
			winQ.pop();

		while (!pseudoSrcQ.empty() && winQ.top().pseudoSrcId < mesh->vertNum &&
			pseudoSrcQ.top().pseudoBirthTime != vertInfos.get(pseudoSrcQ.top().vertID).birthTime)
			pseudoSrcQ.pop();

		if (!winQ.empty() && (pseudoSrcQ.empty() || winQ.top().minDist < pseudoSrcQ.top().dist))
		{
			Window win = winQ.top(); winQ.pop();
			if (win.level > mesh->faceNum) continue;
			// save windows for arbitrary dst geodesic construction
			unsigned twinEdge = mesh->edges[win.edgeID].twinEdge;
			if (twinEdge != -1)
			{
				bool found = false;
				for (int i = 0; i < keptFacesIdx; ++i)
				{
					if (keptFaces[i] == mesh->edges[twinEdge].faceId)
					{
						found = true;
						break;
					}
				}
				if (found && storedWindowsIdx < storedWindowsSize)
					storedWindows[storedWindowsIdx++] = win;
			}
			PropagateWindow(win);
		}
		else if (!pseudoSrcQ.empty() && (winQ.empty() || winQ.top().minDist >= pseudoSrcQ.top().dist))
		{
			PseudoWindow pseudoWin = pseudoSrcQ.top(); pseudoSrcQ.pop();
			if (pseudoWin.level >= mesh->faceNum) continue;
			GenSubWinsForPseudoSrc(pseudoWin);
		}

		if (totalCalcVertNum_ != -1 && totalCalcVertNum >= totalCalcVertNum_)
			break;
	}

	auto vertData = vertInfos.Data();
	unsigned cnt = 0;
	for (int i = 0; i < vertInfos.Size(); ++i)
		if (vertData[i].index != -1) ++cnt;
	cout << "Vertex info hash-table size: " << cnt << "/" << vertInfos.Size() << endl;

	auto splitData = splitInfos.Data();
	cnt = 0;
	for (int i = 0; i < splitInfos.Size(); ++i)
		if (splitData[i].index != -1) ++cnt;
	cout << "Split info hash-table: " << cnt << "/" << splitInfos.Size() << endl;
}

__host__ __device__ void ICH::OutputStatisticInfo()
{
	/*
	cout << "Total generated window number: " << numOfWinGen << endl;
	cout << "Max windows queue size: " << maxWinQSize << endl;
	cout << "Max pseudo-source queue size: " << maxPseudoQSize << endl;
	*/
}

__host__ __device__ double ICH::BuildGeodesicPathTo(unsigned faceId, Vector3D pos, unsigned &srcId,
	unsigned &nextToSrcEdge, double &nextToSrcX, unsigned &nextToDstEdge, double &nextToDstX)
{
	pathPassVert = false;
	// find the window provide the nearest distance
	nextToSrcEdge = -1; nextToDstEdge = -1;
	double minDist = DBL_MAX;
	Window minWin; double xInter; Vector2D pos2D;
	unsigned dstVert = -1;
	bool throughAWindow = true;

	// traverse the surrounded windows
	for (int i = 0; i < storedWindowsIdx; ++i)
	{
		unsigned twinEdge = mesh->edges[storedWindows[i].edgeID].twinEdge;
		if (twinEdge == -1) continue;
		if (mesh->edges[twinEdge].faceId != faceId) continue;

		unsigned e0 = twinEdge;
		unsigned e1 = mesh->edges[e0].nextEdge;
		unsigned e2 = mesh->edges[e1].nextEdge;

		double l0 = mesh->edges[e0].edgeLen;
		double l1 = mesh->edges[e1].edgeLen;
		double l2 = mesh->edges[e2].edgeLen;

		unsigned v0 = mesh->edges[e0].verts[1];
		unsigned v1 = mesh->edges[e0].verts[0];
		unsigned v2 = mesh->edges[e1].verts[1];

		Vector2D p0(0.0, 0.0), p1(l0, 0.0), p2;
		p2.x = (l1*l1 + l0*l0 - l2*l2) / (2.0*l0);
		p2.y = -sqrt(fabs(l1*l1 - p2.x*p2.x));

		// window's pseudo source's 2D planar coordinate
		Vector2D src2D = storedWindows[i].FlatenedSrc();

		// dst point's centroid coordinates
		double a = (pos - mesh->verts[v0].pos).length();
		double b = (pos - mesh->verts[v1].pos).length();
		double c = (pos - mesh->verts[v2].pos).length();

		double s0 = (b + c + l2) / 2.0;
		double s1 = (a + c + l1) / 2.0;
		double s2 = (a + b + l0) / 2.0;

		s0 = sqrt(fabs(s0 * (s0 - b) * (s0 - c) * (s0 - l2)));
		s1 = sqrt(fabs(s1 * (s1 - a) * (s1 - c) * (s1 - l1)));
		s2 = sqrt(fabs(s2 * (s2 - a) * (s2 - b) * (s2 - l0)));

		double w0 = s0 / (s0 + s1 + s2);
		double w1 = s1 / (s0 + s1 + s2);
		double w2 = s2 / (s0 + s1 + s2);

		Vector2D curPos2D = w0 * p0 + w1 * p1 + w2 * p2;

		// calculate the shortest distance
		double curXInter = src2D.x - (curPos2D.x - src2D.x) / (curPos2D.y - src2D.y) * src2D.y;
		double curMinDist = DBL_MAX;
		if (curXInter > storedWindows[i].b0 && curXInter < storedWindows[i].b1)
			curMinDist = (curPos2D - src2D).length() + storedWindows[i].pseudoSrcDist;
		else if (curXInter <= storedWindows[i].b0)
			curMinDist = (curPos2D - Vector2D(storedWindows[i].b0, 0.0)).length() + storedWindows[i].d0 + storedWindows[i].pseudoSrcDist;
		else
			curMinDist = (curPos2D - Vector2D(storedWindows[i].b1, 0.0)).length() + storedWindows[i].d1 + storedWindows[i].pseudoSrcDist;

		if (curMinDist < minDist)
		{
			minDist = curMinDist;
			minWin = storedWindows[i];
			xInter = curXInter;
			pos2D = curPos2D;
		}
	}

	// traverse the surrounded vertices
	for (int i = 0; i < 3; ++i)
	{
		unsigned opVert = mesh->edges[mesh->faces[faceId].edges[i]].verts[0];
		if (mesh->verts[opVert].angle < 2.0 * PI) continue;

		double curDist = (pos - mesh->verts[opVert].pos).length() + vertInfos.get(opVert).dist;
		if (curDist < minDist)
		{
			throughAWindow = false;
			dstVert = opVert;
			minDist = curDist;
		}
	}

	if (minDist == DBL_MAX) {
		return minDist;
	}

	if (!throughAWindow)
	{
		BuildGeodesicPathTo(dstVert, srcId, nextToSrcEdge, nextToSrcX, nextToDstEdge, nextToDstX);
		GeodesicKeyPoint gkp;
		gkp.isVertex = true; gkp.id = dstVert;
		pathPassVert = true;
		nextToDstEdge = mesh->verts[gkp.id].firstEdge; nextToDstX = 0.0;
	}
	else
	{
		// next key point is on an edge
		GeodesicKeyPoint gkp;
		gkp.isVertex = false;
		gkp.id = mesh->edges[minWin.edgeID].twinEdge;
		gkp.pos = mesh->edges[gkp.id].edgeLen - xInter;

		nextToDstEdge = gkp.id; nextToDstX = gkp.pos;

		unsigned enterEdge = gkp.id;
		unsigned opVert = mesh->edges[gkp.id].twinEdge;
		opVert = mesh->edges[mesh->edges[opVert].nextEdge].verts[1];
		double l0 = mesh->edges[gkp.id].edgeLen;
		double l1 = mesh->edges[mesh->edges[gkp.id].nextEdge].edgeLen;
		double l2 = mesh->edges[mesh->edges[mesh->edges[gkp.id].nextEdge].nextEdge].edgeLen;

		Vector2D lastPoint = pos2D, curPoint;
		curPoint.x = l0 - gkp.pos; curPoint.y = 0.0;

		while (minWin.pseudoSrcId < mesh->vertNum && opVert != minWin.pseudoSrcId ||
			minWin.pseudoSrcId >= mesh->vertNum &&
			mesh->edges[mesh->edges[gkp.id].twinEdge].faceId != sourcePointFace)
		{
			// trace back
			unsigned e0 = mesh->edges[gkp.id].twinEdge;
			unsigned e1 = mesh->edges[e0].nextEdge;
			unsigned e2 = mesh->edges[e1].nextEdge;
			double l0 = mesh->edges[e0].edgeLen;
			double l1 = mesh->edges[e1].edgeLen;
			double l2 = mesh->edges[e2].edgeLen;

			Vector2D opVert2D;
			opVert2D.x = (l0*l0 + l2*l2 - l1*l1) / (2.0*l0);
			opVert2D.y = sqrt(fabs(l2*l2 - opVert2D.x*opVert2D.x));

			if (toLeft(opVert2D, lastPoint, curPoint))
			{
				Vector2D p0, p1;
				p0.x = (l2*l2 + l1*l1 - l0*l0) / (2.0*l1);
				p0.y = -sqrt(fabs(l2*l2 - p0.x*p0.x));
				p1.x = l1; p1.y = 0.0;
				Vector2D newlastPoint = gkp.pos / l0 * p0 + (1.0 - gkp.pos / l0) * p1;

				gkp.pos = Intersect(lastPoint, curPoint, Vector2D(l0, 0.0), opVert2D);
				gkp.pos = (1.0 - gkp.pos) * l1;
				gkp.id = e1;
				curPoint.x = l1 - gkp.pos; curPoint.y = 0.0;
				lastPoint = newlastPoint;
			}
			else
			{
				Vector2D p0, p1;
				p0.x = 0.0; p0.y = 0.0;
				p1.x = (l2*l2 + l0*l0 - l1*l1) / (2.0*l2);
				p1.y = -sqrt(fabs(l0*l0 - p1.x*p1.x));
				Vector2D newlastPoint = gkp.pos / l0 * p0 + (1.0 - gkp.pos / l0) * p1;

				gkp.pos = Intersect(lastPoint, curPoint, opVert2D, Vector2D(0.0, 0.0));
				gkp.pos = (1.0 - gkp.pos) * l2;
				gkp.id = e2;
				curPoint.x = l2 - gkp.pos; curPoint.y = 0.0;
				lastPoint = newlastPoint;
			}
			nextToSrcEdge = gkp.id; nextToSrcX = gkp.pos;

			opVert = mesh->edges[gkp.id].twinEdge;
			opVert = mesh->edges[mesh->edges[opVert].nextEdge].verts[1];
		}

		if (minWin.pseudoSrcId >= mesh->vertNum) {
			dstVert = minWin.pseudoSrcId;
			srcId = dstVert;
		}
		else if (vertInfos.get(opVert).dist != 0.0)
		{
			gkp.isVertex = true;
			gkp.id = opVert;
			nextToSrcEdge = mesh->verts[gkp.id].firstEdge; nextToSrcX = 0.0;
			dstVert = opVert;
			pathPassVert = true;
			unsigned nextToDstEdge_; double nextToDstX_;
			BuildGeodesicPathTo(opVert, srcId, nextToSrcEdge, nextToSrcX, nextToDstEdge_, nextToDstX_);
		}
		srcId = minWin.srcID;
	}

	return minDist;
}

__host__ __device__ void ICH::BuildGeodesicPathTo(unsigned vertId, unsigned &srcId,
	unsigned &nextToSrcEdge, double &nextToSrcX, unsigned &nextToDstEdge, double &nextToDstX)
{
	// TODO: build geodesic path from vertex vertId to source
	pathPassVert = false;
	nextToSrcEdge = -1; nextToDstEdge = -1;
	unsigned curVert = vertId;
	GeodesicKeyPoint gkp;
	while (vertInfos.get(curVert).dist != 0.0)
	{
		unsigned enterEdge = vertInfos.get(curVert).enterEdge;
		if (enterEdge == -1)
		{
			// trace back to an arbitrary point
			double curPlanarDist = (sourcePointPos - mesh->verts[curVert].pos).length();
			srcId = mesh->vertNum;
			if (nextToDstEdge == -1)
			{
				nextToDstEdge = mesh->verts[curVert].firstEdge;
				nextToDstX = 0.0;
			}
			else
			{
				nextToSrcEdge = mesh->verts[curVert].firstEdge;
				nextToSrcX = 0.0;
			}
			pathPassVert = true;
			return;
		}
		else if (mesh->edges[enterEdge].verts[0] == curVert)
		{
			// next key point is still a vertex
			unsigned nextVert = mesh->edges[enterEdge].verts[1];
			if (!vertInfos.get(nextVert).isSource)
			{
				gkp.isVertex = true;
				gkp.id = nextVert;

				if (nextToDstEdge == -1)
				{
					nextToDstEdge = mesh->edges[enterEdge].nextEdge;
					nextToDstX = 0.0;
				}
				else
				{
					nextToSrcEdge = mesh->edges[enterEdge].nextEdge;
					nextToSrcX = 0.0;
				}
				pathPassVert = true;
			}
			curVert = nextVert;
		}
		else
		{
			// next key point is on an edge
			gkp.isVertex = false;
			gkp.id = enterEdge; gkp.pos = splitInfos.get(enterEdge).x;

			if (nextToDstEdge == -1)
			{
				nextToDstEdge = gkp.id; nextToDstX = gkp.pos;
			}
			else
			{
				nextToSrcEdge = gkp.id; nextToSrcX = gkp.pos;
			}

			unsigned opVert = mesh->edges[gkp.id].twinEdge;
			opVert = mesh->edges[mesh->edges[opVert].nextEdge].verts[1];
			double l0 = mesh->edges[gkp.id].edgeLen;
			double l1 = mesh->edges[mesh->edges[gkp.id].nextEdge].edgeLen;
			double l2 = mesh->edges[mesh->edges[mesh->edges[gkp.id].nextEdge].nextEdge].edgeLen;

			Vector2D lastPoint, curPoint;
			lastPoint.x = (l1*l1 + l0*l0 - l2*l2) / (2.0*l0);
			lastPoint.y = -sqrt(fabs(l1*l1 - lastPoint.x*lastPoint.x));
			curPoint.x = l0 - gkp.pos; curPoint.y = 0.0;

			while (splitInfos.get(enterEdge).pseudoSrcId < mesh->vertNum &&
				opVert != splitInfos.get(enterEdge).pseudoSrcId ||
				splitInfos.get(enterEdge).pseudoSrcId >= mesh->vertNum &&
				mesh->edges[mesh->edges[gkp.id].twinEdge].faceId != sourcePointFace)
			{
				// trace back
				unsigned e0 = mesh->edges[gkp.id].twinEdge;
				unsigned e1 = mesh->edges[e0].nextEdge;
				unsigned e2 = mesh->edges[e1].nextEdge;
				double l0 = mesh->edges[e0].edgeLen;
				double l1 = mesh->edges[e1].edgeLen;
				double l2 = mesh->edges[e2].edgeLen;

				Vector2D opVert2D;
				opVert2D.x = (l0*l0 + l2*l2 - l1*l1) / (2.0*l0);
				opVert2D.y = sqrt(fabs(l2*l2 - opVert2D.x*opVert2D.x));

				if (toLeft(opVert2D, lastPoint, curPoint))
				{
					Vector2D p0, p1;
					p0.x = (l2*l2 + l1*l1 - l0*l0) / (2.0*l1);
					p0.y = -sqrt(fabs(l2*l2 - p0.x*p0.x));
					p1.x = l1; p1.y = 0.0;
					Vector2D newlastPoint = gkp.pos / l0 * p0 + (1.0 - gkp.pos / l0) * p1;

					gkp.pos = Intersect(lastPoint, curPoint, Vector2D(l0, 0.0), opVert2D);
					gkp.pos = (1.0 - gkp.pos) * l1;
					gkp.id = e1;
					curPoint.x = l1 - gkp.pos; curPoint.y = 0.0;
					lastPoint = newlastPoint;
				}
				else
				{
					Vector2D p0, p1;
					p0.x = 0.0; p0.y = 0.0;
					p1.x = (l2*l2 + l0*l0 - l1*l1) / (2.0*l2);
					p1.y = -sqrt(fabs(l0*l0 - p1.x*p1.x));
					Vector2D newlastPoint = gkp.pos / l0 * p0 + (1.0 - gkp.pos / l0) * p1;

					gkp.pos = Intersect(lastPoint, curPoint, opVert2D, Vector2D(0.0, 0.0));
					gkp.pos = (1.0 - gkp.pos) * l2;
					gkp.id = e2;
					curPoint.x = l2 - gkp.pos; curPoint.y = 0.0;
					lastPoint = newlastPoint;
				}
				nextToSrcEdge = gkp.id; nextToSrcX = gkp.pos;

				opVert = mesh->edges[gkp.id].twinEdge;
				opVert = mesh->edges[mesh->edges[opVert].nextEdge].verts[1];
			}

			if (splitInfos.get(enterEdge).pseudoSrcId >= mesh->vertNum) {
				curVert = splitInfos.get(enterEdge).pseudoSrcId;
				break;
			}
			if (vertInfos.get(opVert).dist != 0.0)
			{
				gkp.isVertex = true;
				gkp.id = opVert;
				nextToSrcEdge = mesh->verts[gkp.id].firstEdge; nextToSrcX = 0.0;
				pathPassVert = true;
			}
			curVert = opVert;
		}
	}
	srcId = curVert;
}

__host__ __device__ double ICH::GetDistanceTo(unsigned vertId)
{
	return vertInfos.get(vertId).dist;
}

__host__ __device__ void ICH::Clear()
{
	winQ.clear(); pseudoSrcQ.clear();

	unsigned size = splitInfos.Size();
	SplitItem *splitData = splitInfos.Data();
	for (int i = 0; i < size + 1; ++i)
	{
		splitData[i].item.dist = DBL_MAX;
		splitData[i].item.x = DBL_MAX;
		splitData[i].item.pseudoSrcId = -1;
		splitData[i].index = -1;
	}

	size = vertInfos.Size();
	VertItem *vertData = vertInfos.Data();
	for (int i = 0; i < size + 1; ++i)
	{
		vertData[i].item.birthTime = -1;
		vertData[i].item.dist = DBL_MAX;
		vertData[i].item.enterEdge = -1;
		vertData[i].item.isSource = false;
		vertData[i].index = -1;
	}
	sourceVert = -1;
	sourcePointFace = -1;

	storedWindowsIdx = 0;
	keptFacesIdx = 0;

	numOfWinGen = 0;
	maxWinQSize = 0;
	maxPseudoQSize = 0;
	totalCalcVertNum = 0;
}

__host__ __device__ void ICH::Initialize()
{
	if (sourceVert != -1)
	{
		int startEdge = mesh->verts[sourceVert].firstEdge;
		int curEdge = startEdge;
		do
		{
			unsigned opEdge = mesh->edges[curEdge].nextEdge;
			Window win;
			win.edgeID = opEdge;
			win.b0 = 0.0; win.b1 = mesh->edges[opEdge].edgeLen;
			win.d0 = mesh->edges[curEdge].edgeLen;
			win.d1 = mesh->edges[mesh->edges[curEdge].prevEdge].edgeLen;
			win.pseudoSrcDist = 0.0; win.calcMinDist();
			win.srcID = sourceVert; win.pseudoSrcId = sourceVert;
			win.pseudoSrcBirthTime = 0;
			win.level = 0;
			winQ.push(win, win.minDist);
			++numOfWinGen;

			unsigned opVert = mesh->edges[curEdge].verts[1];
			vertInfos[opVert].birthTime = 0;
			vertInfos[opVert].dist = mesh->edges[curEdge].edgeLen;
			vertInfos[opVert].enterEdge = mesh->edges[curEdge].twinEdge;

			if (mesh->verts[opVert].angle < 2.0 * PI)
			{
				curEdge = mesh->edges[curEdge].twinEdge;
				if (curEdge != -1) curEdge = mesh->edges[curEdge].nextEdge;
				continue;
			}

			PseudoWindow pseudoWin;
			pseudoWin.vertID = opVert; pseudoWin.dist = mesh->edges[curEdge].edgeLen;
			pseudoWin.srcId = sourceVert; pseudoWin.pseudoSrcId = sourceVert;
			pseudoWin.pseudoBirthTime = vertInfos.get(opVert).birthTime;
			pseudoWin.level = 0;
			pseudoSrcQ.push(pseudoWin, pseudoWin.dist);

			curEdge = mesh->edges[curEdge].twinEdge;
			if (curEdge != -1) curEdge = mesh->edges[curEdge].nextEdge;

		} while (curEdge != startEdge && curEdge != -1);
		vertInfos[sourceVert].birthTime = 0;
		vertInfos[sourceVert].dist = 0.0;
		vertInfos[sourceVert].enterEdge = -1;
		vertInfos[sourceVert].isSource = true;
	}

	if (sourcePointFace != -1)
	{
		for (int j = 0; j < 3; ++j)
		{
			unsigned opEdge = mesh->faces[sourcePointFace].edges[j];
			Window win;
			win.edgeID = opEdge;
			win.b0 = 0.0; win.b1 = mesh->edges[opEdge].edgeLen;
			win.d0 = (sourcePointPos - mesh->verts[mesh->edges[opEdge].verts[0]].pos).length();
			win.d1 = (sourcePointPos - mesh->verts[mesh->edges[opEdge].verts[1]].pos).length();
			win.pseudoSrcDist = 0.0; win.calcMinDist();
			win.srcID = mesh->vertNum; win.pseudoSrcId = win.srcID;
			win.pseudoSrcBirthTime = 0; win.level = 0;
			winQ.push(win, win.minDist);

			unsigned opVert = mesh->edges[opEdge].verts[0];
			vertInfos[opVert].birthTime = 0;
			vertInfos[opVert].dist = (sourcePointPos - mesh->verts[opVert].pos).length();
			vertInfos[opVert].enterEdge = -1;

			if (mesh->verts[opVert].angle < 2.0 * PI) continue;

			PseudoWindow pseudoWin;
			pseudoWin.vertID = opVert;
			pseudoWin.dist = (mesh->verts[opVert].pos - sourcePointPos).length();
			pseudoWin.srcId = win.srcID; pseudoWin.pseudoSrcId = win.srcID;
			pseudoWin.pseudoBirthTime = vertInfos.get(opVert).birthTime;
			pseudoWin.level = 0;
			pseudoSrcQ.push(pseudoWin, pseudoWin.dist);
		}
	}
}

__host__ __device__ void ICH::PropagateWindow(const Window &win)
{
	unsigned e0 = mesh->edges[win.edgeID].twinEdge;
	if (e0 == -1) return;
	unsigned e1 = mesh->edges[e0].nextEdge;
	unsigned e2 = mesh->edges[e1].nextEdge;
	unsigned opVert = mesh->edges[e1].verts[1];

	Vector2D src2D = win.FlatenedSrc();

	Vector2D left(win.b0, 0.0), right(win.b1, 0.0);
	double l0 = mesh->edges[e0].edgeLen;
	double l1 = mesh->edges[e1].edgeLen;
	double l2 = mesh->edges[e2].edgeLen;
	Vector2D v0(0.0, 0.0), v1(l0, 0.0), v2;
	v2.x = (l1*l1 + l0*l0 - l2*l2) / (2.0 * l0);
	v2.y = -sqrt(fabs(l1*l1 - v2.x*v2.x));

	double interX = v2.x - v2.y * (v2.x - src2D.x) / (v2.y - src2D.y);
	Window leftChildWin, rightChildWin;
	bool hasLeftChild = true, hasRightChild = true;
	// only generate right window
	if (interX <= left.x)
	{
		hasLeftChild = false;
		double t0 = Intersect(src2D, left, v2, v1);
		double t1 = Intersect(src2D, right, v2, v1);
		BuildWindow(win, e2, t0, t1, v2, v1, rightChildWin);
		if (!IsValidWindow(rightChildWin, false)) hasRightChild = false;
	}
	// only generate left window
	else if (interX >= right.x)
	{
		hasRightChild = false;
		double t0 = Intersect(src2D, left, v0, v2);
		double t1 = Intersect(src2D, right, v0, v2);
		BuildWindow(win, e1, t0, t1, v0, v2, leftChildWin);
		if (!IsValidWindow(leftChildWin, true)) hasLeftChild = false;
	}
	// generate both left and right window
	else
	{
		double directDist = (v2 - src2D).length();
		// ONE ANGLE, ONE SPLIT
		if (directDist + win.pseudoSrcDist > splitInfos.get(e0).dist &&
			(directDist + win.pseudoSrcDist) / splitInfos.get(e0).dist - 1.0 > RELATIVE_ERROR)
		{
			hasLeftChild = splitInfos.get(e0).x < interX;
			hasRightChild = !hasLeftChild;
			/*cout << "Filter 1 works..." << endl;*/
		}
		else
		{
			splitInfos[e0].dist = directDist + win.pseudoSrcDist;
			splitInfos[e0].pseudoSrcId = win.pseudoSrcId;
			splitInfos[e0].x = l0 - interX;

			if (directDist + win.pseudoSrcDist < vertInfos.get(opVert).dist)
			{
				if (vertInfos.get(opVert).dist == DBL_MAX
					/*&& (win.pseudoSrcId == sourceVert || win.pseudoSrcId == mesh->vertNum)*/)
					++totalCalcVertNum;

				++vertInfos[opVert].birthTime;
				vertInfos[opVert].dist = directDist + win.pseudoSrcDist;
				vertInfos[opVert].enterEdge = e0;
				if (mesh->verts[opVert].angle > 2.0 * PI)
				{
					PseudoWindow pseudoWin;
					pseudoWin.vertID = opVert; pseudoWin.dist = vertInfos.get(opVert).dist;
					pseudoWin.srcId = win.srcID; pseudoWin.pseudoSrcId = win.pseudoSrcId;
					pseudoWin.pseudoBirthTime = vertInfos.get(opVert).birthTime;
					pseudoWin.level = win.level + 1;
					pseudoSrcQ.push(pseudoWin, pseudoWin.dist);
				}
			}
		}
		if (hasLeftChild)
		{
			// left child window
			double t0 = Intersect(src2D, left, v0, v2);
			BuildWindow(win, e1, t0, 0.0, v0, v2, leftChildWin);
			if (!IsValidWindow(leftChildWin, true)) hasLeftChild = false;
		}
		if (hasRightChild)
		{
			// right child window
			double t1 = Intersect(src2D, right, v2, v1);
			BuildWindow(win, e2, 1.0, t1, v2, v1, rightChildWin);
			if (!IsValidWindow(rightChildWin, false)) hasRightChild = false;
		}
	}

	if (hasLeftChild)
	{
		++numOfWinGen;
		winQ.push(leftChildWin, leftChildWin.minDist);
	}
	if (hasRightChild)
	{
		++numOfWinGen;
		winQ.push(rightChildWin, rightChildWin.minDist);
	}

}

__host__ __device__ void ICH::GenSubWinsForPseudoSrc(const PseudoWindow &pseudoWin)
{
	unsigned startEdge, endEdge;
	if (vertInfos.get(pseudoWin.vertID).enterEdge == -1 && vertInfos.get(pseudoWin.vertID).birthTime != -1)
	{
		startEdge = mesh->verts[pseudoWin.vertID].firstEdge;
		endEdge = startEdge;
	}
	else if (mesh->edges[vertInfos.get(pseudoWin.vertID).enterEdge].verts[0] == pseudoWin.vertID)
		GenSubWinsForPseudoSrcFromPseudoSrc(pseudoWin, startEdge, endEdge);
	else if (mesh->edges[mesh->edges[vertInfos.get(pseudoWin.vertID).enterEdge].nextEdge].verts[1] == pseudoWin.vertID)
		GenSubWinsForPseudoSrcFromWindow(pseudoWin, startEdge, endEdge);
	else assert(false);

	// generate windows
	do
	{
		Window win;
		win.edgeID = mesh->edges[startEdge].nextEdge;
		win.b0 = 0.0; win.b1 = mesh->edges[win.edgeID].edgeLen;
		win.d0 = mesh->edges[startEdge].edgeLen;
		win.d1 = mesh->edges[mesh->edges[win.edgeID].nextEdge].edgeLen;
		win.pseudoSrcDist = pseudoWin.dist; win.calcMinDist();
		win.srcID = pseudoWin.srcId; win.pseudoSrcId = pseudoWin.vertID;
		win.pseudoSrcBirthTime = pseudoWin.pseudoBirthTime;
		win.level = pseudoWin.level + 1;
		winQ.push(win, win.minDist);
		++numOfWinGen;

		startEdge = mesh->edges[mesh->edges[mesh->edges[startEdge].nextEdge].nextEdge].twinEdge;
	} while (startEdge != endEdge);

	// generate adjacent pseudo sources
	startEdge = mesh->verts[pseudoWin.vertID].firstEdge;
	int curEdge = startEdge;
	do
	{
		unsigned opVert = mesh->edges[curEdge].verts[1];
		if (mesh->verts[opVert].angle < 2.0 * PI ||
			vertInfos.get(opVert).dist < pseudoWin.dist + mesh->edges[curEdge].edgeLen)
		{
			curEdge = mesh->edges[curEdge].twinEdge;
			if (curEdge != -1) curEdge = mesh->edges[curEdge].nextEdge;
			continue;
		}

		if (vertInfos.get(opVert).dist == DBL_MAX
			/*&& (pseudoWin.pseudoSrcId == sourceVert || pseudoWin.pseudoSrcId == mesh->vertNum)*/)
			++totalCalcVertNum;

		vertInfos[opVert].dist = pseudoWin.dist + mesh->edges[curEdge].edgeLen;
		++vertInfos[opVert].birthTime;
		vertInfos[opVert].enterEdge = mesh->edges[curEdge].twinEdge;

		PseudoWindow childPseudoWin;
		childPseudoWin.vertID = opVert; childPseudoWin.dist = vertInfos.get(opVert).dist;
		childPseudoWin.srcId = pseudoWin.srcId; childPseudoWin.pseudoSrcId = pseudoWin.vertID;
		childPseudoWin.pseudoBirthTime = vertInfos.get(opVert).birthTime;
		childPseudoWin.level = pseudoWin.level;
		pseudoSrcQ.push(childPseudoWin, childPseudoWin.dist);

		curEdge = mesh->edges[curEdge].twinEdge;
		if (curEdge != -1) curEdge = mesh->edges[curEdge].nextEdge;
	} while (curEdge != startEdge && curEdge != -1);
}

__host__ __device__ void ICH::GenSubWinsForPseudoSrcFromWindow(const PseudoWindow &pseudoWin, unsigned &startEdge, unsigned &endEdge)
{
	unsigned e0 = vertInfos.get(pseudoWin.vertID).enterEdge;
	unsigned e1 = mesh->edges[e0].nextEdge;
	unsigned e2 = mesh->edges[e1].nextEdge;

	double l0 = mesh->edges[e0].edgeLen;
	double l1 = mesh->edges[e1].edgeLen;
	double l2 = mesh->edges[e2].edgeLen;

	unsigned pseudoSrc = pseudoWin.vertID;
	Vector2D enterPoint;
	enterPoint.x = l0 - splitInfos.get(e0).x;
	enterPoint.y = 0.0;

	Vector2D v0(0.0, 0.0), v1(l0, 0.0), v2;
	v2.x = (l1*l1 + l0*l0 - l2*l2) / (2.0*l0);
	v2.y = -sqrt(fabs(l1*l1 - v2.x*v2.x));

	// TODO: generate windows using opVert as pseudo sources
	double angle0 = (enterPoint - v2) * (v0 - v2) / (enterPoint - v2).length() / l1;
	double angle1 = (enterPoint - v2) * (v1 - v2) / (enterPoint - v2).length() / l2;
	if (angle0 > 1.0) angle0 = 1.0; else if (angle0 < -1.0) angle0 = -1.0;
	if (angle1 > 1.0) angle1 = 1.0; else if (angle1 < -1.0) angle1 = -1.0;
	angle0 = acos(angle0); angle1 = acos(angle1);

	startEdge = -1, endEdge = -1;
	// traverse from left
	unsigned curEdge = mesh->edges[e1].twinEdge;
	while (angle0 < PI && curEdge != -1)
	{
		unsigned opEdge = mesh->edges[curEdge].nextEdge;
		unsigned nextEdge = mesh->edges[opEdge].nextEdge;
		double L0 = mesh->edges[curEdge].edgeLen, L1 = mesh->edges[nextEdge].edgeLen;
		double L2 = mesh->edges[opEdge].edgeLen;
		double curAngle = (L0*L0 + L1*L1 - L2*L2) / (2.0 * L0 * L1);
		if (curAngle > 1.0) curAngle = 1.0; else if (curAngle < -1.0) curAngle = -1.0;
		curAngle = acos(curAngle);
		angle0 += curAngle;
		curEdge = mesh->edges[nextEdge].twinEdge;
	}
	if (curEdge != -1)
		startEdge = mesh->edges[mesh->edges[curEdge].twinEdge].nextEdge;

	// traverse from right
	curEdge = mesh->edges[e2].twinEdge;
	while (angle1 < PI && curEdge != -1)
	{
		unsigned nextEdge = mesh->edges[curEdge].nextEdge;
		unsigned opEdge = mesh->edges[nextEdge].nextEdge;
		double L0 = mesh->edges[curEdge].edgeLen, L1 = mesh->edges[nextEdge].edgeLen;
		double L2 = mesh->edges[opEdge].edgeLen;
		double curAngle = (L0*L0 + L1*L1 - L2*L2) / (2.0 * L0 * L1);
		if (curAngle > 1.0) curAngle = 1.0; else if (curAngle < -1.0) curAngle = -1.0;
		curAngle = acos(curAngle);
		angle1 += curAngle;
		curEdge = mesh->edges[nextEdge].twinEdge;
	}
	if (curEdge != -1)
	{
		endEdge = mesh->edges[mesh->edges[mesh->edges[curEdge].twinEdge].nextEdge].nextEdge;
		endEdge = mesh->edges[endEdge].twinEdge;
	}
}

__host__ __device__ void ICH::GenSubWinsForPseudoSrcFromPseudoSrc(const PseudoWindow &pseudoWin, unsigned &startEdge, unsigned &endEdge)
{
	unsigned pseudoSrc = pseudoWin.vertID;

	// TODO: generate windows using opVert as pseudo sources
	double angle0 = 0.0, angle1 = 0.0;

	startEdge = -1, endEdge = -1;
	// traverse from left
	unsigned curEdge = vertInfos.get(pseudoWin.vertID).enterEdge;
	while (angle0 < PI && curEdge != -1)
	{
		unsigned opEdge = mesh->edges[curEdge].nextEdge;
		unsigned nextEdge = mesh->edges[opEdge].nextEdge;
		double L0 = mesh->edges[curEdge].edgeLen, L1 = mesh->edges[nextEdge].edgeLen;
		double L2 = mesh->edges[opEdge].edgeLen;
		double curAngle = (L0*L0 + L1*L1 - L2*L2) / (2.0 * L0 * L1);
		if (curAngle > 1.0) curAngle = 1.0; else if (curAngle < -1.0) curAngle = -1.0;
		curAngle = acos(curAngle);
		angle0 += curAngle;
		curEdge = mesh->edges[nextEdge].twinEdge;
	}
	if (curEdge != -1)
		startEdge = mesh->edges[mesh->edges[curEdge].twinEdge].nextEdge;

	// traverse from right
	curEdge = mesh->edges[vertInfos.get(pseudoWin.vertID).enterEdge].twinEdge;
	while (angle1 < PI && curEdge != -1)
	{
		unsigned nextEdge = mesh->edges[curEdge].nextEdge;
		unsigned opEdge = mesh->edges[nextEdge].nextEdge;
		double L0 = mesh->edges[curEdge].edgeLen, L1 = mesh->edges[nextEdge].edgeLen;
		double L2 = mesh->edges[opEdge].edgeLen;
		double curAngle = (L0*L0 + L1*L1 - L2*L2) / (2.0 * L0 * L1);
		if (curAngle > 1.0) curAngle = 1.0; else if (curAngle < -1.0) curAngle = -1.0;
		curAngle = acos(curAngle);
		angle1 += curAngle;
		curEdge = mesh->edges[nextEdge].twinEdge;
	}
	if (curEdge != -1)
	{
		endEdge = mesh->edges[mesh->edges[mesh->edges[curEdge].twinEdge].nextEdge].nextEdge;
		endEdge = mesh->edges[endEdge].twinEdge;
	}
}

__host__ __device__ bool ICH::IsValidWindow(const Window &win, bool isLeftChild)
{
	if (win.b1 <= win.b0) return false;
	// apply ICH's filter
	unsigned v1 = mesh->edges[win.edgeID].verts[0];
	unsigned v2 = mesh->edges[win.edgeID].verts[1];
	unsigned v3 = mesh->edges[mesh->edges[win.edgeID].nextEdge].verts[1];
	double l0 = mesh->edges[win.edgeID].edgeLen;
	double l1 = mesh->edges[mesh->edges[win.edgeID].nextEdge].edgeLen;
	double l2 = mesh->edges[mesh->edges[mesh->edges[win.edgeID].nextEdge].nextEdge].edgeLen;
	Vector2D p1(0.0, 0.0), p2(l0, 0.0), p3;
	p3.x = (l2*l2 + l0*l0 - l1*l1) / (2.0 * l0);
	p3.y = sqrt(fabs(l2*l2 - p3.x*p3.x));

	Vector2D A(win.b0, 0.0), B(win.b1, 0.0);
	Vector2D src2D = win.FlatenedSrc();


	if (win.pseudoSrcDist + (src2D - B).length() > vertInfos.get(v1).dist + win.b1 &&
		(win.pseudoSrcDist + (src2D - B).length()) / (vertInfos.get(v1).dist + win.b1) - 1.0 > 0.0)
	{
		/*cout << "Filter 2 works..." << endl;*/
		return false;
	}
	if (win.pseudoSrcDist + (src2D - A).length() > vertInfos.get(v2).dist + l0 - win.b0 &&
		(win.pseudoSrcDist + (src2D - A).length()) / (vertInfos.get(v2).dist + l0 - win.b0) - 1.0 > 0.0)
	{
		/*cout << "Filter 2 works..." << endl;*/
		return false;
	}
	if (isLeftChild)
	{
		if (win.pseudoSrcDist + (src2D - A).length() > vertInfos.get(v3).dist + (p3 - A).length() &&
			(win.pseudoSrcDist + (src2D - A).length()) / (vertInfos.get(v3).dist + (p3 - A).length()) - 1.0 > 0.0)
		{
			/*cout << "Filter 2 works..." << endl;*/
			return false;
		}
	}
	else
	{
		if (win.pseudoSrcDist + (src2D - B).length() > vertInfos.get(v3).dist + (p3 - B).length() &&
			(win.pseudoSrcDist + (src2D - B).length()) / (vertInfos.get(v3).dist + (p3 - B).length()) - 1.0 > RELATIVE_ERROR)
		{
			/*cout << "Filter 2 works..." << endl;*/
			return false;
		}
	}
	return true;
}

__host__ __device__ void ICH::BuildWindow(const Window &fatherWin,
	unsigned edge,
	double t0, double t1,
	const Vector2D &v0, const Vector2D &v1,
	Window &win)
{
	Vector2D src2D = fatherWin.FlatenedSrc();
	win.edgeID = edge;
	win.b0 = (1 - t0) * mesh->edges[edge].edgeLen; win.b1 = (1 - t1) * mesh->edges[edge].edgeLen;
	win.d0 = (src2D - (t0 * v0 + (1 - t0)*v1)).length();
	win.d1 = (src2D - (t1 * v0 + (1 - t1)*v1)).length();
	win.pseudoSrcDist = fatherWin.pseudoSrcDist;
	win.calcMinDist();
	win.srcID = fatherWin.srcID; win.pseudoSrcId = fatherWin.pseudoSrcId;
	win.pseudoSrcBirthTime = fatherWin.pseudoSrcBirthTime;
	win.level = fatherWin.level + 1;
}

__host__ __device__ double ICH::Intersect(const Vector2D &v0, const Vector2D &v1, const Vector2D &p0, const Vector2D &p1)
{
	double a00 = p0.x - p1.x, a01 = v1.x - v0.x, b0 = v1.x - p1.x;
	double a10 = p0.y - p1.y, a11 = v1.y - v0.y, b1 = v1.y - p1.y;
	return (b0*a11 - b1*a01) / (a00*a11 - a10*a01);
}