#include "hip/hip_runtime.h"
#include "SVG.cuh"
#include "book.cuh"
#include <iostream>
#include <fstream>

using namespace std;

__global__ void constructSVG(Mesh mesh,
	SVG::PQWinItem *d_winPQs, SVG::PQPseudoWinItem *d_pseudoWinPQs,
	ICH::SplitInfo *d_splitInfoBuf, ICH::VertInfo *d_vertInfoBuf)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int totalThreadNum = blockDim.x * gridDim.x;
	int vertPerThread = (mesh.vertNum + totalThreadNum - 1) / totalThreadNum;

	PriorityQueues<ICH::Window> winPQ;
	PriorityQueues<ICH::PseudoWindow> pseudoWinPQ;
	winPQ.AssignMemory(d_winPQs + idx * WIN_PQ_SIZE); 
	pseudoWinPQ.AssignMemory(d_pseudoWinPQs + idx * PSEUDOWIN_PQ_SIZE);

	ICH ich;
	ich.AssignMesh(&mesh); 
	ich.AssignBuffers(d_splitInfoBuf + idx * mesh.edgeNum, 
		d_vertInfoBuf + idx * mesh.vertNum, 
		winPQ, pseudoWinPQ);

	for (int i = idx; i < mesh.vertNum; i += vertPerThread)
	{
		// TODO: run ICH
		ich.Clear();
		ich.AddSource(i);
		ich.Execute();
		break;
	}
}

SVG::SVG()
{

}

SVG::~SVG()
{

}

void SVG::AssignMesh(Mesh *mesh_, Mesh *d_mesh_)
{
	mesh = mesh_; d_mesh = d_mesh_;
}

bool SVG::Allocation()
{
	// allocation memories for PriorityQueues

	int totalThreadNum = THREAD_NUM * BLOCK_NUM; 

	hipError_t cudaStatus;

	HANDLE_ERROR(hipMalloc((void**)&d_winPQs, totalThreadNum * WIN_PQ_SIZE * sizeof(PQWinItem)));
	HANDLE_ERROR(hipMalloc((void**)&d_pseudoWinPQs, totalThreadNum * PSEUDOWIN_PQ_SIZE * sizeof(PQPseudoWinItem)));

	// allocation info buffers for ICH
	HANDLE_ERROR(hipMalloc((void**)&d_splitInfoBuf, totalThreadNum * mesh->edgeNum * sizeof(ICH::SplitInfo)));
	HANDLE_ERROR(hipMalloc((void**)&d_vertInfoBuf, totalThreadNum * mesh->vertNum * sizeof(ICH::VertInfo)));

	return true;
}

void SVG::ConstructSVG()
{
	constructSVG <<<BLOCK_NUM, THREAD_NUM >>>(*d_mesh, d_winPQs, d_pseudoWinPQs, d_splitInfoBuf, d_vertInfoBuf);
	// TODO: organize the constructed SVG
}